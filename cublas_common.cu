#include <stdio.h>
#include <stdlib.h>
#include "cublas_common.h"

const char* cublasGetErrorString(hipblasStatus_t status){
	switch(status){
	case HIPBLAS_STATUS_SUCCESS: return "HIPBLAS_STATUS_SUCCESS";
	case HIPBLAS_STATUS_NOT_INITIALIZED: return "HIPBLAS_STATUS_NOT_INITIALIZED";
	case HIPBLAS_STATUS_ALLOC_FAILED: return "HIPBLAS_STATUS_ALLOC_FAILED";
	case HIPBLAS_STATUS_INVALID_VALUE: return "HIPBLAS_STATUS_INVALID_VALUE"; 
	case HIPBLAS_STATUS_ARCH_MISMATCH: return "HIPBLAS_STATUS_ARCH_MISMATCH"; 
	case HIPBLAS_STATUS_MAPPING_ERROR: return "HIPBLAS_STATUS_MAPPING_ERROR";
	case HIPBLAS_STATUS_EXECUTION_FAILED: return "HIPBLAS_STATUS_EXECUTION_FAILED"; 
	case HIPBLAS_STATUS_INTERNAL_ERROR: return "HIPBLAS_STATUS_INTERNAL_ERROR"; 
	default: return "unknown error";
	}
}
void cuBlasStatusError( hipblasStatus_t err,const char *file,int line ) {
	if (err != HIPBLAS_STATUS_SUCCESS) {
		printf( "cuBLAS Error:\n%s in %s at line %d\n", cublasGetErrorString(err),file, line );
		exit( EXIT_FAILURE );
	}
}

