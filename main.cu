#include <iostream>
#include "cuda_common.h"
#include "cublas_common.h"
#include "matrix_array.h"

const int calc = 100;

#ifndef __CUBLAS_XT__
void gemm(hipblasHandle_t cublas,mtk::MatrixXf& A,mtk::MatrixXf &B, mtk::MatrixXf &C){
	float alpha = 1.0f,beta = 0.0f;
	CUBLAS_HANDLE_ERROR( hipblasSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
				A.getRows(),B.getCols(),A.getRows(),
				&alpha,
				A.getDevicePointer(),A.getRows(),
				B.getDevicePointer(),B.getRows(),
				&beta,
				C.getDevicePointer(),C.getRows()) );
}
#else
void gemm(cublasXtHandle_t cublas,mtk::MatrixXf& A,mtk::MatrixXf &B, mtk::MatrixXf &C){
	float alpha = 1.0f,beta = 0.0f;
	CUBLAS_HANDLE_ERROR( cublasXtSgemm(cublas,HIPBLAS_OP_N,HIPBLAS_OP_N,
				A.getRows(),B.getCols(),A.getRows(),
				&alpha,
				A.getDevicePointer(),A.getRows(),
				B.getDevicePointer(),B.getRows(),
				&beta,
				C.getDevicePointer(),C.getRows()) );
}
#endif

void testSgemm(int dim){
	std::cout<<dim<<" x "<<dim<<std::endl;
	mtk::MatrixXf A,B,C;
	A.setSize(dim,dim)->allocateDevice()->initDeviceRandom(-1.0f,1.0f);
	B.setSize(dim,dim)->allocateDevice()->initDeviceRandom(-1.0f,1.0f);
	C.setSize(dim,dim)->allocateDevice()->initDeviceRandom(-1.0f,1.0f);

#ifndef __CUBLAS_XT__
	hipblasHandle_t cublas;
	CUBLAS_HANDLE_ERROR( hipblasCreate( &cublas ) );
#else
	cublasXtHandle_t cublas;
	CUBLAS_HANDLE_ERROR( cublasXtCreate( &cublas ));
	int devices[] = {0,1,2,3};
	CUBLAS_HANDLE_ERROR(cublasXtDeviceSelect(cublas,1,devices));
	CUBLAS_HANDLE_ERROR(cublasXtSetBlockDim(cublas,2) );
#endif
	hipEvent_t start,stop;
	float elapsed_time;
	CUDA_HANDLE_ERROR(hipEventCreate(&start));
	CUDA_HANDLE_ERROR(hipEventCreate(&stop));
	// cache
	gemm(cublas,A,B,C);
	CUDA_HANDLE_ERROR( hipEventRecord(start) );
	for(int c = 0;c < calc;c++){
		gemm(cublas,A,B,C);
	}
	CUDA_HANDLE_ERROR( hipEventRecord( stop ) );
	CUDA_HANDLE_ERROR( hipEventSynchronize( stop ));
	CUDA_HANDLE_ERROR( hipEventElapsedTime( &elapsed_time, start, stop ));
	std::cout<<"hipblasSgemm : "<<(elapsed_time/calc)<<" ms"<<std::endl;
	CUDA_HANDLE_ERROR( hipEventDestroy( start ) );
	CUDA_HANDLE_ERROR( hipEventDestroy( stop ) );
#ifndef __CUBLAS_XT__
	CUBLAS_HANDLE_ERROR( hipblasDestroy( cublas ) );
#else
	CUBLAS_HANDLE_ERROR( cublasXtDestroy( cublas ));
#endif
}

int main(){
	for(int i=4;i<16;i++)
		testSgemm(1<<i);
}
